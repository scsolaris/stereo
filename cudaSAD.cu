#include "hip/hip_runtime.h"
#include "cudaSAD.h"

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <stdio.h>
   
#define SQ(a) (__mul24(a,a));

extern int divUp(int a, int b);

static int g_w;
static int g_h;

static float *g_disparityLeft;
static int *g_minSAD;
static size_t g_floatDispPitch;

static hipArray * g_leftTex_array;
static hipArray * g_rightTex_array;

static unsigned int LeftImage_GLBufferID;
static unsigned int RightImage_GLBufferID;
static unsigned int DisparityImage_GLBufferID;

static texture<unsigned char, 2, hipReadModeNormalizedFloat> leftTex;
static texture<unsigned char, 2, hipReadModeNormalizedFloat> rightTex;

#define ROWSperTHREAD 40 // the number of rows a thread will process
#define BLOCK_W 64 // the thread block width
#define RADIUS_H 5 // Kernel Radius 5V & 5H = 11x11 kernel
#define RADIUS_V 5
#define MIN_SAD 500000 // The mimium acceptable SAD value
#define STEREO_MIND 0.0f // The minimum d range to check
#define STEREO_DISP_STEP 1.0f // the d step, must be <= 1 to avoid aliasing
#define SHARED_MEM_SIZE ((BLOCK_W + 2*RADIUS_H)*sizeof(int) ) // amount of 

__global__ void stereoKernel( float *disparityPixel,int *disparityMinSAD,int width,int height,size_t out_pitch,float STEREO_MAXD); 

void SetupStereo(unsigned int w, unsigned int h)
{
	g_w = w;
	g_h = h;

	hipMallocPitch((void**)&g_disparityLeft,&g_floatDispPitch,w*sizeof(float),h);
	hipMallocPitch((void**)&g_minSAD,&g_floatDispPitch,w*sizeof(int),h);
	g_floatDispPitch /= sizeof(float);

	hipChannelFormatDesc U8Tex = hipCreateChannelDesc<unsigned char>();
	hipMallocArray(&g_leftTex_array, &U8Tex, g_w, g_h);
	hipMallocArray(&g_rightTex_array, &U8Tex, g_w, g_h);

//	size_t free, total;
//	hipMemGetInfo(&free,&total);
//	printf("Memory After Allocation - Free: %d, Total: %d\n",free/(1024*1024),total/(1024*1024));
}

float cudaSAD(unsigned char * p_hostLeft, unsigned char * p_hostRight,unsigned char *Dispmap,int w,int h,float MAXD,int scale)
{
	SetupStereo(w,h);

	unsigned int timer;
	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));

	dim3 grid(1,1,1);
	dim3 threads(16,8,1);

	// Greyscale Image, just copy it.
	hipMemcpyToArray(g_leftTex_array, 0, 0, p_hostLeft,g_w * g_h,
	hipMemcpyHostToDevice);
	hipMemcpyToArray(g_rightTex_array, 0, 0, p_hostRight,g_w * g_h,
	hipMemcpyHostToDevice);

	// Set up the texture parameters for bilinear interpolation & clamping
	leftTex.filterMode = hipFilterModeLinear;
	hipBindTextureToArray(leftTex, g_leftTex_array);
	rightTex.filterMode = hipFilterModeLinear;
	hipBindTextureToArray(rightTex, g_rightTex_array);

	threads.x = BLOCK_W;
	threads.y = 1;
	grid.x = divUp(g_w, BLOCK_W);
	grid.y = divUp(g_h,ROWSperTHREAD);
	stereoKernel<<<grid,threads,SHARED_MEM_SIZE>>>(g_disparityLeft,g_minSAD,
	g_w,g_h,g_floatDispPitch,MAXD);

	hipDeviceSynchronize();
	hipUnbindTexture(leftTex);
	hipUnbindTexture(rightTex);
	CUT_SAFE_CALL(cutStopTimer(timer)); // don't time the drawing
	float retval = cutGetTimerValue(timer);

	float* tDispmap = new float[g_floatDispPitch*g_h];
	if(hipSuccess!=hipMemcpy2D(tDispmap,sizeof(float)*g_w,g_disparityLeft,sizeof(float)*g_floatDispPitch,g_w*sizeof(float),g_h,hipMemcpyDeviceToHost)) printf("wrong!\n");
	for(int i=0;i<g_h;i++)
		for(int j=0;j<g_w;j++)
		{
//			printf("%f",tDispmap[i*g_floatDispPitch+j]);
			Dispmap[i*g_w+j] = (unsigned char)(tDispmap[i*g_w+j]*scale);
		}

	delete []tDispmap;
	return retval;	
}

__global__ void stereoKernel( float *disparityPixel,int *disparityMinSAD,int width,int height,size_t out_pitch,float STEREO_MAXD)
{
	extern __shared__ int col_sad[]; // column squared difference functions
	float d; // disparity value
	int diff; // difference temporary value
	int sad; // total SAD for a kernel
	float x_tex; // texture coordinates for image lookup
	float y_tex;
	int row; // the current row in the rolling window
	int i; // for index variable

	// use define¡¯s to save registers
	#define X (__mul24(blockIdx.x,BLOCK_W) + threadIdx.x)
	#define Y (__mul24(blockIdx.y,ROWSperTHREAD))

	// for threads reading the extra border pixels, this is the offset
	// into shared memory to store the values
	int extra_read_val = 0;
	if(threadIdx.x < (2*RADIUS_H)) extra_read_val = BLOCK_W+threadIdx.x;
	// initialize the memory used for the disparity and the disparity difference
	if(X<width )
	{
		for(i = 0;i < ROWSperTHREAD && Y+i < height;i++)
		{
			disparityPixel[__mul24((Y+i),out_pitch)+X] = -1;
//			disparityDiff[__mul24((Y+i),out_pitch)+X] = MIN_DISP;
			disparityMinSAD[__mul24((Y+i),out_pitch)+X] = MIN_SAD;
		}
	}
	__syncthreads();

	if( X < (width+RADIUS_H) && Y <= (height) )
	{
		x_tex = X - RADIUS_H;
		for(d = STEREO_MIND; d <= STEREO_MAXD; d += STEREO_DISP_STEP)
		{
			col_sad[threadIdx.x] = 0;
			if(extra_read_val>0) col_sad[extra_read_val] = 0;
			// do the first rows
			y_tex = Y - RADIUS_V;
			for(i = 0; i <= 2*RADIUS_V; i++)
			{
				diff = (int)(255.0f*tex2D(leftTex,x_tex,y_tex)) - (int)(255.0f*tex2D(rightTex,x_tex-d,y_tex));
				col_sad[threadIdx.x] += SQ(diff);
				if(extra_read_val > 0)
				{
					diff = (int)(255.0f*tex2D(leftTex,x_tex+BLOCK_W,y_tex)) - (int)(255.0f*tex2D(rightTex,x_tex+BLOCK_W-d,y_tex));
					col_sad[extra_read_val] += SQ(diff);
				}
				y_tex += 1.0f;
			}
			__syncthreads();
			// now accumulate the total
			if(X < width && Y < height)
			{
				sad = 0;
				for(i = 0;i<(2*RADIUS_H);i++)
				{
					sad += col_sad[i+threadIdx.x];
				}
				if(sad < disparityMinSAD[__mul24(Y,out_pitch) + X])
				{
					disparityPixel[__mul24(Y,out_pitch) + X] = d;
					disparityMinSAD[Y*out_pitch + X] = sad;
				}
			}
			__syncthreads();
			// now do the remaining rows
			y_tex = (float)(Y - RADIUS_V); // this is the row we will remove
			for(row = 1;row < ROWSperTHREAD && (row+Y < (height+RADIUS_V)); row++)
			{
				// subtract the value of the first row from column sums
				diff = (int)(255.0f*tex2D(leftTex,x_tex,y_tex)) - (int)(255.0f*tex2D(rightTex,x_tex-d,y_tex));
				col_sad[threadIdx.x] -= SQ(diff);
				// add in the value from the next row down
				diff = (int)(255.0f*tex2D(leftTex,x_tex,y_tex + (float)(2*RADIUS_V)+1.0f)) -
					(int)(255.0f*tex2D(rightTex,x_tex-d,y_tex +(float)(2*RADIUS_V)+1.0f));
				col_sad[threadIdx.x] += SQ(diff);
				if(extra_read_val > 0)
				{
					diff = (int)(255.0f*tex2D(leftTex,x_tex+(float)BLOCK_W,y_tex)) -
					(int)(255.0f*tex2D(rightTex,x_tex-d+(float)BLOCK_W,y_tex));
					col_sad[threadIdx.x+BLOCK_W] -= SQ(diff);
					diff = (int)(255.0f*tex2D(leftTex,x_tex+(float)BLOCK_W,y_tex +
					(float)(2*RADIUS_V)+1.0f)) -
					(int)(255.0f*tex2D(rightTex,x_tex-d+(float)BLOCK_W,y_tex +
					(float)(2*RADIUS_V)+1.0f));
					col_sad[extra_read_val] += SQ(diff);
				}
				y_tex += 1.0f;
				__syncthreads();
				if(X<width && (Y+row) < height)
				{
					sad = 0;
					for(i = 0;i<(2*RADIUS_H);i++)
					{
						sad += col_sad[i+threadIdx.x];
					}
					if(sad < disparityMinSAD[__mul24(Y+row,out_pitch) + X])
					{
						disparityPixel[__mul24(Y+row,out_pitch) + X] = d;
						disparityMinSAD[__mul24(Y+row,out_pitch) + X] = sad;
					}
				}
				__syncthreads(); // wait for everything to complete
			} // for row loop
		} // for d loop
	} // if 'int the image' loop
}
